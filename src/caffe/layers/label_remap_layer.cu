#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>
#include <cstdio>

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {
template <typename Dtype>
__global__ void LabelRemapForward(const int n, const int* label_map_data,
    const Dtype* in, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = static_cast<Dtype>(
        label_map_data[static_cast<int>(in[index])]);
    //printf("******%d %d\n", static_cast<int>(in[index]),
    //    label_map_data[static_cast<int>(in[index])]);
  }
}

template <typename Dtype>
void LabelRemapLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  const int* label_map_data = (int*) label_map->gpu_data();
  // NOLINT_NEXT_LINE(whitespace/operators)
  LabelRemapForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count, label_map_data, bottom_data, top_data);
  CUDA_POST_KERNEL_CHECK;
}

INSTANTIATE_LAYER_GPU_FORWARD(LabelRemapLayer);


}  // namespace caffe
